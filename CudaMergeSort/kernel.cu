#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
using namespace std;

__global__ 
void singleMergeSort(int * array, int size)
{
	//Block size is given by size
    int thread = threadIdx.x;
	int block = blockIdx.x;

	int leftStart = thread * size;
	int rightStart = thread * size + size / 2;

	//make another block to track the numbers
	int * fullBlock;
	//malloc(&fullBlock, sizeof(int) * size);
	fullBlock = new int[size];
	int tracker = 0;
	//printf("Left Num: %d, Right Num: %d, Thread Num: %d, Size: %d\n", leftStart, rightStart, thread, size);
	while (leftStart < thread * size + size / 2 && rightStart < (thread + 1) * (size))
	{
		//Figure out an inplace method for this later?

		//If left side is less than or left side is completely full
		if (array[leftStart] > array[rightStart])
		{
			fullBlock[tracker] = array[rightStart];
			rightStart++;
		}
		else
		{
			fullBlock[tracker] = array[leftStart];
			leftStart++;
		}
		tracker++;
	}

	while (leftStart < thread * size + size / 2)
	{
		fullBlock[tracker++] = array[leftStart++];
	}
	while (rightStart < (thread + 1) * (size))
	{
		fullBlock[tracker++] = array[rightStart++];
	}
	printf("Left Num: %d, Right Num: %d, Thread Num: %d, Size: %d\n", leftStart, rightStart, thread, size);
	memcpy(&(array[thread * size]), fullBlock, sizeof(int) * size);
	free(fullBlock);

	//printf("Left Num: %d, Right Num: %d, Thread Num: %d\n", array[thread * size], array[thread * size + size / 2], thread);
	//printf("TwoVals: %d, %d\n", array[thread], array[thread + 1]);
	//printf("Thread ID: %d\n\tBlock ID: %d\n", thread, block);

}

void printArray(int * array, int size)
{
	for (int i = 0; i < size; i++)
	{
		cout << array[i] << ' ';
	}
	cout << endl << endl;
}

void sortArray(int * array, int size)
{
	//Copy the array to device code first
	int * deviceArray;
	hipMalloc(&deviceArray, size * sizeof(int));
	hipMemcpy(deviceArray, array, size * sizeof(int), hipMemcpyHostToDevice);

	//Print original array first
	printArray(array, size);
	//first iteration of this code, size must be a factor of 2
	int sizeSort = 2;
	while (sizeSort <= size)
	{
		int numBlocks = size / sizeSort;
		int numThreads = numBlocks;
		cout << sizeSort << endl;

		singleMergeSort << <1, numBlocks >> > (deviceArray, sizeSort);
		hipDeviceSynchronize();
		hipMemcpy(array, deviceArray, size * sizeof(int), hipMemcpyDeviceToHost);
		
		printArray(array, size);
		sizeSort *= 2;
	}

	hipFree(deviceArray);

}

int main()
{
	//Generate array and its values for testing purposes
	int * sortThis = new int[16];
	
	sortThis[0] = 6;
	sortThis[1] = 3;
	sortThis[2] = 2;
	sortThis[3] = 14;
	sortThis[4] = 28;
	sortThis[5] = 1;
	sortThis[6] = 4;
	sortThis[7] = 9;
	sortThis[8] = 57;
	sortThis[9] = 77;
	sortThis[10] = 28;
	sortThis[11] = 22;
	sortThis[12] = 22;
	sortThis[13] = 213;
	sortThis[14] = 5;
	sortThis[15] = 0;
	sortThis[16] = 3;

	sortArray(sortThis, 16);
	//delete []sortThis;
	return 0;
}
